#include "hip/hip_runtime.h"
/* Each kernel handles the update of one pagerank score. In other
 * words, each kernel handles one row of the update:
 *
 *      pi(t+1) = (1/2) A pi(t) + (1 / (2N))
 *      
 * You may assume that num_nodes <= blockDim.x * 65535
 *
 */
__global__
void device_graph_propagate(const uint* graph_indices
		, const uint* graph_edges
		, const float* graph_nodes_in
		, float* graph_nodes_out
		, const float* inv_edges_per_node
		, int num_nodes) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < num_nodes){
		float sum = 0.f;

		//for all of its edges
		for(uint j = graph_indices[i]; j < graph_indices[i+1]; j++) {
			sum += graph_nodes_in[ graph_edges[j] ] * inv_edges_per_node[ graph_edges[j] ];
		}

		graph_nodes_out[i] = 0.5f/(float)num_nodes + 0.5f*sum;
	}
}

/* This function executes a specified number of iterations of the
 * pagerank algorithm. The variables are:
 *
 * h_graph_indices, h_graph_edges:
 *     These arrays describe the indices of the neighbors of node i.
 *     Specifically, node i is adjacent to all nodes in the range
 *     h_graph_edges[h_graph_indices[i] ... h_graph_indices[i+1]].
 *
 * h_node_values_input:
 *     An initial guess of pi(0).
 *
 * h_gpu_node_values_output:
 *     Output array for the pagerank vector.
 *
 * h_inv_edges_per_node:
 *     The i'th element in this array is the reciprocal of the
 *     out degree of the i'th node.
 *
 * nr_iterations:
 *     The number of iterations to run the pagerank algorithm for.
 *
 * num_nodes:
 *     The number of nodes in the whole graph (ie N).
 *
 * avg_edges:
 *     The average number of edges in the graph. You are guaranteed
 *     that the whole graph has num_nodes * avg_edges edges.
 *
 */


double device_graph_iterate(const uint* h_graph_indices
		, const uint* h_graph_edges
		, const float* h_node_values_input
		, float* h_gpu_node_values_output
		, const float* h_inv_edges_per_node
		, int nr_iterations
		, int num_nodes
		, int avg_edges) {

	// TODO: allocate GPU memory
	size_t size = num_nodes * sizeof(float);
	float* d_buffer_1;
	hipMalloc(&d_buffer_1, size);
	float* d_buffer_2;
	hipMalloc(&d_buffer_2, size);
	uint* d_graph_indices;
	hipMalloc(&d_graph_indices, (num_nodes+1) * sizeof(int));
	uint* d_graph_edges;
	hipMalloc(&d_graph_edges, num_nodes * avg_edges * sizeof(int));
	float* d_inv_edges_per_node;
	hipMalloc(&d_inv_edges_per_node, size);

	// TODO: check for allocation failure

	// TODO: copy data to the GPU
	hipMemcpy(d_buffer_1, h_node_values_input, size, hipMemcpyHostToDevice);
	hipMemcpy(d_graph_indices, h_graph_indices, (num_nodes+1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_graph_edges, h_graph_edges, num_nodes * avg_edges * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_inv_edges_per_node, h_inv_edges_per_node , size, hipMemcpyHostToDevice);

	start_timer(&timer);

	const int block_size = 256;
	const int bnam = num_nodes / block_size + 1;

	
	// TODO: launch your kernels the appropriate number of iterations
	for(int iter = 0; iter < nr_iterations / 2; iter++) {
		device_graph_propagate<<<bnam,block_size>>>(d_graph_indices, d_graph_edges, d_buffer_1, d_buffer_2,
				d_inv_edges_per_node, num_nodes);
		device_graph_propagate<<<bnam,block_size>>>(d_graph_indices, d_graph_edges, d_buffer_2, d_buffer_1,
				d_inv_edges_per_node, num_nodes);
	}
	if(nr_iterations % 2)
		device_graph_propagate<<<bnam,block_size>>>(d_graph_indices, d_graph_edges, d_buffer_1, d_buffer_2,
				d_inv_edges_per_node, num_nodes);

	// This two line below is original code.
	check_launch("gpu graph propagate");
	double gpu_elapsed_time = stop_timer(&timer);

	// TODO: copy final data back to the host for correctness checking
	// handle the odd case and copy memory to the output location
	if(nr_iterations % 2) {
		hipMemcpy(h_gpu_node_values_output, d_buffer_2, size, hipMemcpyDeviceToHost);
	} else {
		hipMemcpy(h_gpu_node_values_output, d_buffer_1, size, hipMemcpyDeviceToHost);
	}

	// TODO: free the memory you allocated!
	hipFree(d_buffer_1);
	hipFree(d_buffer_2);
	hipFree(d_graph_indices);
	hipFree(d_graph_edges);
	hipFree(d_inv_edges_per_node);

	check_launch("gpu hipFree function");

	return gpu_elapsed_time;
}
